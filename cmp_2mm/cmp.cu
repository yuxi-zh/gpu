#include "utils.h"
#include "stdio.h"
#include "stdlib.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"

#include <vector>
#include <iostream>
#include <iomanip>

inline void random_fill(float *A, int size) {
	for (int i = 0; i < size; ++i)
		A[i] = rand() / (float)RAND_MAX;
}

void time_evaluator(int m, int n, int l, bool ta, bool tb) {

	hipblasHandle_t handle;
	float *T= NULL, alpha = 1.0f, beta = 0.0f;
	
	CHECK_CUBLAS_CALL(hipblasCreate(&handle));
	
	float *HA, *HB, *HC;
	ASSERT((HA = (float *)malloc(sizeof(float) * (l * n))) != NULL);
	ASSERT((HB = (float *)malloc(sizeof(float) * (l * m))) != NULL);
	ASSERT((HC = (float *)malloc(sizeof(float) * (m * n))) != NULL);
	
	random_fill(HA, (l * n));
	random_fill(HB, (l * m));

	float *DA, *DB, *DC;
	CHECK_CUDA_CALL(hipMalloc((void **)&DA, sizeof(float) * (l * n)));
	CHECK_CUDA_CALL(hipMalloc((void **)&DB, sizeof(float) * (l * m)));
	CHECK_CUDA_CALL(hipMalloc((void **)&DC, sizeof(float) * (m * n)));

	CHECK_CUBLAS_CALL(hipblasSetVector((l * n), sizeof(float), HA, 1, DA, 1));
	CHECK_CUBLAS_CALL(hipblasSetVector((l * m), sizeof(float), HB, 1, DB, 1));
	
	CHECK_CUBLAS_CALL(hipblasCreate(&handle));
	CHECK_CUDA_CALL(hipMalloc((void **)&T, sizeof(float) * n * n));

	hipblasOperation_t opa = ta ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	hipblasOperation_t opb = tb ? HIPBLAS_OP_T : HIPBLAS_OP_N;

	int lda = ta ? l : m;
	int ldb = tb ? n : l;

	hipEvent_t start, end;
	float sum = 0.0, tmp;
	int num_runs = 10;
	for (int _ = 0; _ < num_runs; _++)
	{
		CHECK_CUDA_CALL(hipEventCreate(&start));
		CHECK_CUDA_CALL(hipEventCreate(&end));
		CHECK_CUDA_CALL(hipEventRecord(start, 0));
		CHECK_CUBLAS_CALL(hipblasSgemm(
			handle, opa, opb, m, n, l, &alpha, DA, lda, DB, ldb, &beta, DC, m
		));
		CHECK_CUDA_CALL(hipEventRecord(end, 0));
		CHECK_CUDA_CALL(hipEventSynchronize(end));
		CHECK_CUDA_CALL(hipEventElapsedTime(&tmp, start, end));
		CHECK_CUDA_CALL(hipEventDestroy(start));
		CHECK_CUDA_CALL(hipEventDestroy(end));
		sum += tmp;
	}
	float t = sum / num_runs;
	float GFLOPS = (2.0 * m * n * l) / t / 1e6;
	std::cout << std::boolalpha << ta << '\t';
	std::cout << std::boolalpha << tb << '\t';
    printf("%4d\t%4d\t%4d\t%.8g\t%g\n", m, n, l, t, GFLOPS);


	CHECK_CUBLAS_CALL(hipblasDestroy(handle));

	CHECK_CUDA_CALL(hipFree(DA));
	CHECK_CUDA_CALL(hipFree(DB));
	CHECK_CUDA_CALL(hipFree(DC));	
	
	free(HA);
	free(HB);
	free(HC);
}

int main(int argc, char const *argv[])
{
	int n_device = 0;
	CHECK_CUDA_CALL(hipGetDeviceCount(&n_device));
	ASSERT_MSG(n_device == 1, "Only consider one device case");

	hipDeviceProp_t device_prop;
	CHECK_CUDA_CALL(hipGetDeviceProperties(&device_prop, 0));
	ASSERT_MSG((device_prop.major << 4) + device_prop.minor >= 0x35,
		"Device API is not supported when cc <= 3.5");

	bool tas[] = {true, false}, tbs[] = {false, true};

	for (int i = 0; i < 2; i++) {
		for (int j = 0; j < 2; j++) {
			for (int scale = 64; scale <= 2048; scale *= 2) {
				time_evaluator(scale, scale, scale, tas[i], tbs[j]);
			}
		}
	}

	return 0;
}