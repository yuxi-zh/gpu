#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "errno.h"
#include "string.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"
// #include "hip/hip_runtime_api.h"

#define N 			(1024)
#define SIZE 		(N * N)
#define OUTER_RUNS	(50)

#define allocate_host_memory(name, size, errhanlde) 							\
	name = (float *)malloc(sizeof(float) * size); 						\
	if (name == NULL) 														\
	{																			\
		fprintf(stderr, "Host memory allocation for %s failed: %s\n",			\
			#name, strerror(errno));											\
		goto errhanlde;															\
	}

#define allocate_device_memory(name, size, errhanlde)							\
	if (hipMalloc((void **)&name, size * sizeof(name[0])) != hipSuccess)		\
	{																			\
		fprintf(stderr, "Device memory allocation for %s failed\n", #name);		\
		goto errhanlde;															\
	}

#define initialize_device_matrices(host, device, size, errhanlde)				\
	status = hipblasSetVector(size, sizeof(host[0]), host, 1, device, 1);		\
	if (status != HIPBLAS_STATUS_SUCCESS)										\
	{																			\
		fprintf(stderr, "device access from %s to %s error\n", #host, #device);	\
		goto errhanlde;															\
	}

#define free_device_memory(name, errhanlde)										\
	if (hipFree(name) != hipSuccess) {								\
		fprintf(stderr, "Memory free %s failed\n", #name);						\
		goto errhanlde;															\
	}

#define random_fill(M, n)														\
	for (int i = 0; i < n; ++i)													\
		M[i] = rand() / (float)RAND_MAX;										\

void threeMatrixMulV0(int n, const float* A, const float* B,
					 const float* C, float* D)
{
	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			D[i * n + j] = 0;
			for (int k = 0; k < n; ++k)
			{
				for (int l = 0; l < n; ++l)
				{
					D[i * n + j] += A[i *n + k] * B[k * n + l] * C[l * n + j];
				}
			}
		}
	}
}

void threeMatrixMulV1(int n, const float* A, const float* B, 
					 const float* C, float* D)
{
	hipblasHandle_t hanlde;
	hipblasStatus_t status = hipblasCreate(&hanlde);
	float *T;
	float alpha = 1.0f, beta = 0.0f;

	if (status != HIPBLAS_STATUS_SUCCESS)
		goto finish;

	allocate_device_memory(T, SIZE, clean_handle);

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, B, n, C, n, &beta, T, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, T, n, &beta, D, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

clean_t:
	free_device_memory(T, clean_handle);

clean_handle:
	hipblasDestroy(hanlde);

finish: return;
}

__global__ void __threeMatrixMulV2(hipblasStatus_t *returnValue, int n,
								 const float* A, const float* B,
								 const float* C, float* D)
{
	hipblasHandle_t hanlde;
	hipblasStatus_t status = hipblasCreate(&hanlde);
	float *T;
	float alpha = 1.0f, beta = 0.0f;

	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_handle;

	T = (float *)malloc(sizeof(float) * SIZE);
	if (T == NULL)
		goto clean_handle;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, B, n, C, n, &beta, T, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, T, n, &beta, D, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

clean_t:
	free(T);

clean_handle:
	hipblasDestroy(hanlde);

	*returnValue = status;
}

void threeMatrixMulV2(int n, const float* A, const float* B, 
						const float* C, float* D)
{
	hipblasStatus_t status;
	hipblasHandle_t handle;

	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CUBLAS initialization failed\n");
		goto finish;
	}

	hipblasStatus_t *dev_status;
	allocate_device_memory(dev_status, 1, cds);		

	__threeMatrixMulV2<<<1, 1>>>(dev_status, N, A, B, C, D);

	hipError_t error;
	if ((error = hipGetLastError()) != hipSuccess)
	{
		fprintf(stderr, "cuda kernel execution failed: %s\n",
			hipGetErrorString(error));
		goto cds;
	}

	if ((error = hipMemcpy(&status, dev_status, sizeof(hipblasStatus_t),
		hipMemcpyDeviceToHost)) != hipSuccess)
	{
		fprintf(stderr, "Device to host memory copy failed: %s\n",
			hipGetErrorString(error));
		goto cds;
	}

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CUBLAS device API call failed: %d\n", status);
		goto cds;
	}

cds:free_device_memory(dev_status, chandle);

chandle:hipblasDestroy(handle);

finish:return;

}

typedef void (*threeMatrixMulFunc)(int, const float*, const float*, 
									const float*, float*);

void __evaluate(threeMatrixMulFunc threeMatrixMul, int n, const float* A,
			 const float* B, const float* C, float* D)
{
	hipEvent_t start, end;
	float sum = 0.0, tmp;
	for (int _ = 0; _ < OUTER_RUNS; _++)
	{
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);
		threeMatrixMul(n, A, B, C, D);
		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&tmp, start, end);
		hipEventDestroy(start);
		hipEventDestroy(end);
		sum += tmp * 1000.0;
	}

	printf("%f\n", sum/OUTER_RUNS);
}

#define evaluate(threeMatrixMul, n, A, B, C, D)									\
	printf("Evalute %s : ", #threeMatrixMul);									\
	__evaluate(threeMatrixMul, n, A, B, C, D);										

int main(int argc, char const *argv[])
{
	hipblasStatus_t status;
	hipblasHandle_t handle;

	int n_device = 0;
	hipGetDeviceCount(&n_device);

	if (n_device != 1)
	{
		fprintf(stderr, "Device count = %d\n", n_device);
		goto finish;
	}

	hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, 0);

	if ((device_prop.major << 4) + device_prop.minor < 0x35)
	{
		printf("Device API is not supported when CC <= 3.5\n");
		goto finish;
	}

	status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        goto chandle;
    }

	float *HA, *HB, *HC, *HD;
	allocate_host_memory(HA, SIZE, finish);
	allocate_host_memory(HB, SIZE, cha);
	allocate_host_memory(HC, SIZE, chb);
	allocate_host_memory(HD, SIZE, chc);

	random_fill(HA, SIZE);
	random_fill(HB, SIZE);
	random_fill(HC, SIZE);

	float *DA, *DB, *DC, *DD;
	allocate_device_memory(DA, SIZE, chd);
	allocate_device_memory(DB, SIZE, cda);
	allocate_device_memory(DC, SIZE, cdb);
	allocate_device_memory(DD, SIZE, cdc);

	initialize_device_matrices(HA, DA, SIZE, cdd);
	initialize_device_matrices(HB, DB, SIZE, cdd);
	initialize_device_matrices(HC, DC, SIZE, cdd);
	initialize_device_matrices(HD, DD, SIZE, cdd);

	evaluate(threeMatrixMulV1, N, DA, DB, DC, DD);
	evaluate(threeMatrixMulV2, N, DA, DB, DC, DD);

cdd:free_device_memory(DD, finish);
cdc:free_device_memory(DC, finish);
cdb:free_device_memory(DB, finish);
cda:free_device_memory(DA, finish);

chd:free(HD);
chc:free(HC);
chb:free(HB);
cha:free(HA);

chandle:hipblasDestroy(handle);

finish:return 0;

}