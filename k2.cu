#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "errno.h"
#include "string.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime_api.h"

#define N 			(1024)
#define SIZE 		(N * N)
#define OUTER_RUNS	(50)

#define allocate_host_memory(name, size, errhanlde) 							\
	float *name = (float *)malloc(sizeof(float) * size); 						\
	if (name == nullptr) 														\
	{																			\
		fprintf(stderr, "Host memory allocation for %s failed: %s\n",			\
			#name, strerrno(errno));											\
		goto errhanlde;															\
	}

#define allocate_device_memory(name, size, errhanlde)							\
	if (hipMalloc((void **)&name, size * sizeof(name[0]) != hipSuccess)		\
	{																			\
		fprintf(stderr, "Device memory allocation for %s failed\n", #name);		\
		goto errhanlde;															\
	}

#define initialize_device_matrices(host, device, size, errhanlde)				\
	status = hipblasSetVector(size, sizeof(host[0]), host, 1, device, 1)			\
	if (status != HIPBLAS_STATUS_SUCCESS)										\
	{																			\
		fprintf(stderr, "device access from %s to %s error\n", #host, #device);	\
		goto errhanlde;															\
	}

#define free_device_memory(name, errhanlde)										\
	if ((error = hipFree(name)) != hipSuccess) {								\
		fprintf(stderr, "Memory free %s failed\n", #name);						\
		goto errhanlde;															\
	}

#define random_fill(M, n)														\
	for (int i = 0; i < n; ++i)													\
		M[i] = rand() / (float)RAND_MAX;										\

void threeMatrixMulV0(int n, const float *A, const float *B,
						const float *C, const float *D)
{
	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			D[i * n + j] = 0;
			for (int k = 0; k < n; ++k)
			{
				for (int l = 0; l < n; ++l)
				{
					D[i * n + j] += A[i *n + k] * B[k * n + l] * C[l * n + j];
				}
			}
		}
	}
}

void threeMatrixMulV1(hipblasStatus_t *returnValue, int n, 
						const float *A, const float *B, 
						const float *C, const float *D)
{
	cublasHanlde_t hanlde;
	hipblasStatus_t status = hipblasCreate(&hanlde);

	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_handle;

	float *DT;
	allocate_device_memory(DT, SIZE, clean_handle);

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 1.0f, B, n, C, n, 0.0f, T, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 1.0f, A, n, T, n, 0.0f, D, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

clean_t:
	free_device_memory(DT);

clean_handle:
	cublasDestory(hanlde);

	*returnValue = status;
}

__global__ void __threeMatrixMulV2(hipblasStatus_t *returnValue, int n,
									const float *A, const float *B,
									const float *C, const float *D)
{
	cublasHanlde_t hanlde;
	hipblasStatus_t status = hipblasCreate(&hanlde);

	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_handle;

	float *T = (float *)malloc(sizeof(float) * SIZE);
	if (T == nullptr)
		goto clean_handle;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 1.0f, B, n, C, n, 0.0f, T, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 1.0f, A, n, T, n, 0.0f, D, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto clean_t;

clean_t:
	free(T);

clean_handle:
	cublasDestory(hanlde);

	*returnValue = status;
}

void threeMatrixMulV2(int n, const float *A, const float *B, 
						const float *C, const float *D)
{
	hipblasStatus_t status;
	cublasHanlde_t handle;

	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CUBLAS initialization failed\n");
		goto finish;
	}

	hipblasStatus_t *dev_status;
	allocate_device_memory(dev_status, 1, cds);		

	threeMatrixMulCuda8<<<1, 1>>>(dev_status, N, DA, DB, DC, DD);

	hipError_t error;
	if ((error = hipGetLastError()) != hipSuccess)
	{
		fprintf(stderr, "cuda kernel execution failed: %s\n",
			hipGetErrorString(error));
		goto cds;
	}

	if ((error = cudaMemCpy(&status, dev_status, sizeof(hipblasStatus_t),
		hipMemcpyDeviceToHost)) != hipSuccess)
	{
		fprintf(stderr, "Device to host memory copy failed: %s\n",
			hipGetLastError(error));
		goto cds;
	}

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CUBLAS device API call failed: %d\n", status);
		goto cds;
	}

	if ((error = cudaMemCpy(&HD, DD, sizeof(float) * SIZE,
		hipMemcpyDeviceToHost)) != hipSuccess)
	{
		fprintf(stderr, "Device to host memory copy failed: %s\n",
			hipGetLastError(error));
		goto cds;
	}

cds:free_device_memory(dev_status, finish);

chandle:cublasDestory(hanlde);

finish:return;

}

typedef void (*threeMatrixMulFunc)(int, const float, const float,
									const float, const float);

void __evalute(threeMatrixMulFunc threeMatrixMul, int n, const float *A,
				const float *B, const float *C, const float *D)
{
	hipEvent_t start, end;
	float sum = 0.0, tmp;
	for (int _ = 0; + < OUTER_RUNS; _++)
	{
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);
		threeMatrixMul(n, A, B, C, D);
		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&tmp, start, end);
		hipEventDestroy(start);
		hipEventDestroy(end);
		sum += tmp * 1000.0;
	}

	printf("%f\n", sum/OUTER_RUNS);
}

#define evalute(threeMatrixMul, n, A, B, C, D)									\
	printf("Evalute %s : ", #threeMatrixMul);									\
	evalute(threeMatrixMul, n, A, B, C, D);										

int main(int argc, char const *argv[])
{
	hipblasStatus_t status;
	cublasHanlde_t hanlde;

	int dev_id = findCudaDevice(argc, (const char **)arv);
	hipDeviceProp_t device_prop;
	checkCudaErrors(hipGetDeviceProperties(&device_prop, device_id));

	if ((device_prop.major << 4) + device_prop.minor < 0x35)
	{
		printf("Device API is not supported when CC <= 3.5\n");
		goto finish;
	}

	status = hipblasCreate(&handle);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        goto chandle;
    }

	float *HA, *HB, *HC, *HD;
	allocate_host_memory(HA, SIZE, finish);
	allocate_host_memory(HB, SIZE, cha);
	allocate_host_memory(HC, SIZE, chb);
	allocate_host_memory(HD, SIZE, chc);

	random_fill(HA, n);
	random_fill(HB, n);
	random_fill(HC, n);

	float *DA, *DB, *DC, *DD;
	allocate_device_memory(DA, SIZE, chd);
	allocate_device_memory(DB, SIZE, cda);
	allocate_device_memory(DC, SIZE, cdb);
	allocate_device_memory(DD, SIZE, cdc);

	initialize_device_matrices(HA, DA, SIZE, cdd);
	initialize_device_matrices(HB, DB, SIZE, cdd);
	initialize_device_matrices(HC, DC, SIZE, cdd);
	initialize_device_matrices(HD, DD, SIZE, cdd);

	evalute(threeMatrixMulV1, N, DA, DB, DC, DD);
	evalute(threeMatrixMulV2, N, DA, DB, DC, DD);

cdd:free_device_memory(DD, finish);
cdc:free_device_memory(DC, finish);
cdb:free_device_memory(DB, finish);
cda:free_device_memory(DA, finish);

chd:free(HD);
chc:free(HC);
chb:free(HB);
cha:free(HA);

chandle:cublasDestory(hanlde);

finish:return 0;

}