#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "errno.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime_api.h"

__global__ void threeMatrixMulCuda8(hipblasStatus_t *returnValue, int n,
									const float *A, const float *B,
									const float *C, const float *D)
{
	cublasHanlde_t hanlde;
	hipblasStatus_t status = hipblasCreate(&hanlde);

	if (status != HIPBLAS_STATUS_SUCCESS)
		goto finish;

	float *T = (float *)malloc(sizeof(float) * n * n);
	if (T == nullptr)
		goto finish;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 1.0f, B, n, C, n, 0.0f, D, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto finish;

	status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 1.0f, A, n, T, n, 0.0f, D, n
	);
	if (status != HIPBLAS_STATUS_SUCCESS)
		goto finish;

finish:
	cublasDestory(hanlde);
	*returnValue = status;
}

#define allocate_host_memory(name, size, errhanlde) 							\
	float *name = (float *)malloc(sizeof(float) * size); 						\
	if (name == nullptr) 														\
	{																			\
		fprintf(stderr, "Host memory allocation for %s failed: %s\n",			\
			#name, strerrno(errno));											\
		goto errhanlde;															\
	}

#define allocate_device_memory(name, size, errhanlde)							\
	if (hipMalloc((void **)&name, size * sizeof(name[0]) != hipSuccess)		\
	{																			\
		fprintf(stderr, "Device memory allocation for %s failed\n", #name);		\
		goto errhanlde;															\
	}

#define initialize_device_matrices(host, device, size, errhanlde)				\
	status = hipblasSetVector(size, sizeof(host[0]), host, 1, device, 1)			\
	if (status != HIPBLAS_STATUS_SUCCESS)										\
	{																			\
		fprintf(stderr, "device access from %s to %s error\n", #host, #device);	\
		goto errhanlde;															\
	}

#define free_device_memory(name, errhanlde)										\
	if ((error = hipFree(name)) != hipSuccess) {								\
		fprintf(stderr, "Memory free %s failed\n", #name);						\
		goto errhanlde;															\
	}

#define random_fill(M, n)														\
	for (int i = 0; i < n; ++i)													\
		M[i] = rand() / (float)RAND_MAX;										\

int main(int argc, char const *argv[])
{
	hipblasStatus_t status;
	cublasHanlde_t hanlde;

	int dev_id = findCudaDevice(argc, (const char **)arv);
	hipDeviceProp_t device_prop;
	checkCudaErrors(hipGetDeviceProperties(&device_prop, device_id));

	if ((device_prop.major << 4) + device_prop.minor < 0x35)
	{
		printf("Device API is not supported when CC <= 3.5\n");
		goto finish;
	}

	float *HA, *HB, *HC, *HD;
	allocate_host_memory(HA, SIZE, finish);
	allocate_host_memory(HB, SIZE, finish);
	allocate_host_memory(HC, SIZE, finish);
	allocate_host_memory(HD, SIZE, finish);

	random_fill(HA, n);
	random_fill(HB, n);
	random_fill(HC, n);

	float *DA, *DB, *DC, *DD;
	allocate_device_memory(DA, SIZE, finish);
	allocate_device_memory(DB, SIZE, finish);
	allocate_device_memory(DC, SIZE, finish);
	allocate_device_memory(DD, SIZE, finish);

	initialize_device_matrices(HA, DA, SIZE, finish);
	initialize_device_matrices(HB, DB, SIZE, finish);
	initialize_device_matrices(HC, DC, SIZE, finish);
	initialize_device_matrices(HD, DD, SIZE, finish);

	hipblasStatus_t *dev_status;
	allocate_device_memory(dev_status, 1, finish);		

	threeMatrixMulCuda8<<<1, 1>>>(dev_status, N, DA, DB, DC, DD);

	hipError_t error;
	if ((error = hipGetLastError()) != hipSuccess)
	{
		fprintf(stderr, "cuda kernel execution failed: %s\n",
			hipGetErrorString(error));
		goto finish;
	}

	if ((error = cudaMemCpy(&status, dev_status, sizeof(hipblasStatus_t),
		hipMemcpyDeviceToHost)) != hipSuccess)
	{
		fprintf(stderr, "Device to host memory copy failed: %s\n",
			hipGetLastError(error));
		goto finish;
	}

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "CUBLAS device API call failed: %d\n", status);
		goto finish;
	}

	if ((error = cudaMemCpy(&HD, DD, sizeof(float) * SIZE,
		hipMemcpyDeviceToHost)) != hipSuccess)
	{
		fprintf(stderr, "Device to host memory copy failed: %s\n",
			hipGetLastError(error));
		goto finish;
	}

	free_device_memory(dev_status, finish);
	free_device_memory(DA, finish);
	free_device_memory(DB, finish);
	free_device_memory(DC, finish);
	free_device_memory(DD, finish);

	return 0;
}