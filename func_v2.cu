#include "hip/hip_runtime.h"

#include "utils.h"
#include "hipblas.h"
#include "hip/hip_runtime.h"

__global__
void __func_v2__(hipblasStatus_t *status, int n, const float* A, const float* B, 
					const float* C, float* D)
{
	hipblasHandle_t hanlde;	
	float *T = NULL, alpha = 1.0f, beta = 0.0f;

	*status = hipblasCreate(&hanlde);
	if (*status != HIPBLAS_STATUS_SUCCESS)
		return;

	T = (float *)malloc(sizeof(float) * n * n);
	if (T == NULL) {
		*status = HIPBLAS_STATUS_ALLOC_FAILED;
		return;
	}

	*status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, B, n, C, n, &beta, T, n
	);
	if (*status != HIPBLAS_STATUS_SUCCESS)
		return;

	*status = hipblasSgemm(
		hanlde, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, T, n, &beta, D, n
	);
	if (*status != HIPBLAS_STATUS_SUCCESS)
		return;

	free(T);
	*status = hipblasDestroy(hanlde);
}

void func_v2(int n, const float* A, const float* B, const float* C, float* D)
{
	hipblasHandle_t handle;
	CHECK_CUBLAS_CALL(hipblasCreate(&handle));

	hipblasStatus_t status;
	hipblasStatus_t *dev_status;
	CHECK_CUDA_CALL(hipMalloc((void **)&dev_status, sizeof(hipblasStatus_t)));

	__func_v2__<<<1, 1>>>(dev_status, n, A, B, C, D);
	CHECK_CUDA_CALL(hipGetLastError());

	CHECK_CUDA_CALL(hipMemcpy(&status, dev_status, sizeof(hipblasStatus_t), 
		hipMemcpyDeviceToHost));
//	ASSERT_MSG(status == HIPBLAS_STATUS_SUCCESS, "CUBLAS device API call failed");
	CHECK_CUBLAS_CALL(status);
	CHECK_CUDA_CALL(hipFree(dev_status));
	CHECK_CUBLAS_CALL(hipblasDestroy(handle));
}
