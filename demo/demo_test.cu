#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "assert.h"
#include "utils.h"

extern __device__ void DeviceCpy(float *A, float *B, int n);

__global__ void GlobalCpy(float *A, float *B, int n) {
    DeviceCpy(A, B, n);
}


int main(int argc, char const *argv[]) {
	
	float *DA = NULL;
	float *DB = NULL;
	CHECK_CUDA_CALL(hipMalloc((void **)&DA, sizeof(float) * 256));	
	CHECK_CUDA_CALL(hipMalloc((void **)&DB, sizeof(float) * 256));

	float *HA = new float[256];
	float *HB = new float[256];
	
	for (int i = 0; i < 256; i++)
		HA[i] = i;
	
	CHECK_CUDA_CALL(hipMemcpy(DA, HA, sizeof(float) * 256, hipMemcpyHostToDevice));
	GlobalCpy<<<1, 256>>>(DA, DB, 256);
	CHECK_CUDA_CALL(hipGetLastError());
	CHECK_CUDA_CALL(hipMemcpy(HB, DB, sizeof(float) * 256, hipMemcpyDeviceToHost));

	for (int i = 0; i < 256; i++)
		assert(HB[i] == i);

	CHECK_CUDA_CALL(hipFree(DA));
	CHECK_CUDA_CALL(hipFree(DB));
	
	delete[] HA;
	delete[] HB;

	return 0;
}
