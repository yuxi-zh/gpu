#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "utils.h"

#include <tuple>
#include <bitset>
#include <vector>
#include <algorithm>
#include <stdexcept>

using namespace std;

extern __device__ void sconv_direct_fprop_128x128(        
	float* param_Sum,
    float* param_X,
    float* param_O,
    float* param_I,
    float* param_F,
    float param_alpha,
    float param_beta,
    unsigned param_flags,
    unsigned param_N,
    unsigned param_K,
    unsigned param_D,
    unsigned param_H,
    unsigned param_W,
    unsigned param_WN,
    unsigned param_HWN,
    unsigned param_DHWN,
    unsigned param_C,
    unsigned param_KRST,
    unsigned param_RST,
    unsigned param_RS,
    unsigned param_T,
    unsigned param_R,
    unsigned param_S,
    unsigned param_magic_RS,
    unsigned param_shift_RS,
    unsigned param_magic_S,
    unsigned param_shift_S,
    int param_pad_d,
    int param_pad_h,
    int param_pad_w,
    unsigned param_str_d,
    unsigned param_str_h,
    unsigned param_str_w,
    unsigned param_dil_d,
    unsigned param_dil_h,
    unsigned param_dil_w,
    unsigned param_P2,
    unsigned param_Q,
    unsigned param_PQk,
    unsigned param_Qk,
    unsigned param_k,
    unsigned param_magic_PQk,
    unsigned param_shift_PQk,
    unsigned param_magic_Qk,
    unsigned param_shift_Qk,
    unsigned param_magic_k,
    unsigned param_shift_k,
    unsigned param_QN,
    unsigned param_PQN,
    unsigned param_MPQN,
    unsigned param_gridN,
    unsigned param_gridQN,
    unsigned param_gridPQN,
    unsigned param_gridMPQN);

__global__ void sconv_direct_fprop_128x128_global(
	float* param_Sum,
    float* param_X,
    float* param_O,
    float* param_I,
    float* param_F,
    float param_alpha,
    float param_beta,
    unsigned param_flags,
    unsigned param_N,
    unsigned param_K,
    unsigned param_D,
    unsigned param_H,
    unsigned param_W,
    unsigned param_WN,
    unsigned param_HWN,
    unsigned param_DHWN,
    unsigned param_C,
    unsigned param_KRST,
    unsigned param_RST,
    unsigned param_RS,
    unsigned param_T,
    unsigned param_R,
    unsigned param_S,
    unsigned param_magic_RS,
    unsigned param_shift_RS,
    unsigned param_magic_S,
    unsigned param_shift_S,
    int param_pad_d,
    int param_pad_h,
    int param_pad_w,
    unsigned param_str_d,
    unsigned param_str_h,
    unsigned param_str_w,
    unsigned param_dil_d,
    unsigned param_dil_h,
    unsigned param_dil_w,
    unsigned param_P2,
    unsigned param_Q,
    unsigned param_PQk,
    unsigned param_Qk,
    unsigned param_k,
    unsigned param_magic_PQk,
    unsigned param_shift_PQk,
    unsigned param_magic_Qk,
    unsigned param_shift_Qk,
    unsigned param_magic_k,
    unsigned param_shift_k,
    unsigned param_QN,
    unsigned param_PQN,
    unsigned param_MPQN,
    unsigned param_gridN,
    unsigned param_gridQN,
    unsigned param_gridPQN,
    unsigned param_gridMPQN) 
{
	sconv_direct_fprop_128x128(
		param_Sum,
    	param_X,
    	param_O,
    	param_I,
    	param_F,
    	param_alpha,
    	param_beta,
    	param_flags,
    	param_N,
    	param_K,
    	param_D,
    	param_H,
    	param_W,
    	param_WN,
    	param_HWN,
    	param_DHWN,
    	param_C,
    	param_KRST,
    	param_RST,
    	param_RS,
    	param_T,
    	param_R,
    	param_S,
    	param_magic_RS,
    	param_shift_RS,
    	param_magic_S,
    	param_shift_S,
    	param_pad_d,
    	param_pad_h,
    	param_pad_w,
    	param_str_d,
    	param_str_h,
    	param_str_w,
    	param_dil_d,
    	param_dil_h,
    	param_dil_w,
    	param_P2,
    	param_Q,
    	param_PQk,
    	param_Qk,
    	param_k,
    	param_magic_PQk,
    	param_shift_PQk,
    	param_magic_Qk,
    	param_shift_Qk,
    	param_magic_k,
    	param_shift_k,
    	param_QN,
    	param_PQN,
    	param_MPQN,
    	param_gridN,
    	param_gridQN,
    	param_gridPQN,
    	param_gridMPQN);
}

unsigned ceil_div(unsigned x, unsigned y)
{
	return -(-x / y);
}

unsigned closest_divisor(unsigned val, unsigned div)
{
	vector<pair<signed, signed> > list;
	for (int i = 1; i < 8; i++) {
		if (val % i == 0) {
			list.push_back(pair<signed, signed>(i - div, -div));
		}
	}
	sort(list.begin(), list.end());
	return -(list[0].second);
}

tuple<unsigned, unsigned> magic32(unsigned nmax, unsigned d)
{
	unsigned nc = ((nmax + 1) / d) * d - 1;
	unsigned nbits = bitset<32>(nmax).to_string().size() - 2;
	for (int p = 0; p < 2 * nbits - 2; p++) {
		if ((1 << p) > nc * (d - 1 - (1 << (p - 1)) % d)) {
			unsigned m = ((1 << p) + d - 1 - (1 << (p - 1)) % d);
			return tuple<unsigned, unsigned>(m, p);
		}
	}
	throw runtime_error("Can't find magic number for division");
}

tuple<unsigned, unsigned> magic64(unsigned d)
{
	unsigned nmax, magic, shift;

	if (d == 3) 
		nmax = 0xffffffff;
	else
		nmax = 0x7fffffff;

	tie(magic, shift) = magic32(nmax, d);
	if (magic != 1)
		shift -= 32;
	return tuple<unsigned, unsigned>(magic, shift);
}

int main(int argc, char const *argv[])
{
    unsigned N = 128;
    unsigned C = 4;
    unsigned K = 64;
    unsigned D = 1;
    unsigned H = 128;
    unsigned W = 128;
    unsigned T = 1;
    unsigned R = 3;
    unsigned S = 3;
    int pad_d = 0;
    int pad_h = 0;
    int pad_w = 0;
    unsigned str_d = 1;
    unsigned str_h = 1;
    unsigned str_w = 1;
    unsigned dil_d = 1;
    unsigned dil_h = 1;
    unsigned dil_w = 1;
    unsigned M = 1;
    unsigned P = (H + 2 * pad_h - (dil_h * (R - 1) + 1)) / str_h + 1;
    unsigned Q = (W + 2 * pad_w - (dil_w * (S - 1) + 1)) / str_w + 1;

    float* Sum = NULL;
    float* X = NULL;
    float* O = new float[N * M * P * Q * K];
    float* I = new float[N * D * H * W * C];
    float* F = new float[K * T * R * S * C];
    float alpha = 1.0;
    float beta = 0.0;
    unsigned flags = 0;

    float* DO = NULL;
    float* DI = NULL;
    float* DF = NULL;

    CHECK_CUDA_CALL(hipMalloc((void **)&DO, sizeof(float) * N * M * P * Q * K));
    CHECK_CUDA_CALL(hipMalloc((void **)&DI, sizeof(float) * N * D * H * W * C));
    CHECK_CUDA_CALL(hipMalloc((void **)&DF, sizeof(float) * K * T * R * S * C));

    CHECK_CUDA_CALL(hipMemcpy(DI, I, sizeof(float) * N * D * H * W * C, hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(DF, F, sizeof(float) * K * T * R * S * C, hipMemcpyHostToDevice));

    unsigned blockK = 128;
    unsigned blockN = 128;

    unsigned gridK = ceil_div(K, blockK);
    unsigned gridN = ceil_div(N, blockN);
    
    unsigned RS = R * S;
    unsigned RST = T * RS;
    unsigned KRST = K * RST;

    unsigned k = closest_divisor(gridK, 128 / blockK);

    unsigned P2 = P / 2;
    unsigned Q2 = Q * 2;
    unsigned Qk = Q2 * k;
    unsigned PQk = P * Q * k;
    
    unsigned magic_PQk, shift_PQk;
    tie(magic_PQk, shift_PQk) = magic64(PQk);
    unsigned magic_Qk, shift_Qk;
    tie(magic_Qk, shift_Qk) = magic64(Qk);
    unsigned magic_k, shift_k;
    tie(magic_k, shift_k) = magic32(Qk, k);
    unsigned magic_RS, shift_RS;
    tie(magic_RS, shift_RS) = magic32(RST + 32, RS);
    unsigned magic_S, shift_S;
    tie(magic_S, shift_S) = magic32(RS + 32, S);

	unsigned bsum_warps = blockN / 64;
    unsigned gridNw = gridN * bsum_warps;
    unsigned gridQNw = Q * gridNw;
    unsigned gridPQNw = P * gridQNw;
	unsigned gridMPQNw = M * gridPQNw;
	unsigned gridMPQ = M * P * Q;    

	dim3 grid(gridMPQ * k, gridK / k, gridN);
	dim3 block(256, 1, 1);

    sconv_direct_fprop_128x128_global<<<grid, block>>>(
    	Sum,
    	X,
    	DO,
    	DI,
    	DF,
    	alpha,
    	beta,
    	flags,
    	N,
    	K,
    	D,
    	H,
    	W,
    	W * N,
    	H * W * N,
    	D * H * W * N,
    	C,
    	KRST,
    	RST,
    	RS,
    	T,
    	R,
    	S,
    	magic_RS,
    	shift_RS,
    	magic_S,
    	shift_S,
    	pad_d,
    	pad_h,
    	pad_w,
    	str_d,
    	str_h,
    	str_w,
    	dil_d,
    	dil_h,
    	dil_w,
    	P2,
    	Q,
    	PQk,
    	Qk,
    	k,
    	magic_PQk,
    	shift_PQk,
    	magic_Qk,
    	shift_Qk,
    	magic_k,
    	shift_k,
    	Q * N,
    	P * Q * N,
    	M * P * Q * N,
    	gridNw,
    	gridQNw,
    	gridPQNw,
    	gridMPQNw);
    CHECK_CUDA_CALL(hipGetLastError());
    CHECK_CUDA_CALL(hipMemcpy(O, DO, sizeof(float) * N * M * P * Q * K, hipMemcpyDeviceToHost));

    CHECK_CUDA_CALL(hipFree(DI));
    CHECK_CUDA_CALL(hipFree(DO));
    CHECK_CUDA_CALL(hipFree(DF));

    delete[] I;
    delete[] F;
    delete[] O;

	return 0;
}